#include "hip/hip_runtime.h"
#include "CUDA.hpp"
#include "MathFunction.hpp"

//------------------------------------------------------------------------------
// ray gen program - the actual rendering happens in here
//------------------------------------------------------------------------------
extern "C" __global__ void __raygen__renderFrame()
{
    // compute a test pattern based on pixel ID
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const int accumID = optixLaunchParams.frame.accumID;
    const auto &camera = optixLaunchParams.camera;

    PRD prd;
    prd.random.init(ix + accumID * optixLaunchParams.frame.size.x,
                    iy + accumID * optixLaunchParams.frame.size.y);
    prd.pixelColor = vec3f(0.f);

    // the values we store the PRD pointer in:
    uint32_t u0, u1;
    packPointer(&prd, u0, u1);

    int numPixelSamples = NUM_PIXEL_SAMPLES;

    vec3f pixelColor = 0.f;
    for (int sampleID = 0; sampleID < numPixelSamples; sampleID++)
    {
        vec3f rayDir;
        if (optixLaunchParams.camera.camera_type == PINHOLE)
        {
            // normalized screen plane position, in [0,1]^2
            const vec2f screen(vec2f(ix + prd.random() - 0.5f, iy + prd.random() - 0.5f) / vec2f(optixLaunchParams.frame.size));

            // generate ray direction
            rayDir = normalize(camera.direction + (screen.x - 0.5f) * camera.horizontal + (screen.y - 0.5f) * camera.vertical);
        }
        else if (optixLaunchParams.camera.camera_type == ENV)
        {
            // sperical coordinate position
            vec3f spherical_position((ix + prd.random() - 0.5f) * camera.horizontal + (iy + prd.random() - 0.5f) * camera.vertical);
            spherical_position -= vec3f(0.f, M_PI, 0.f);
            // change into xyz coordinate position
            const vec3f xyz_position(sphere_to_normal(spherical_position));
            // view port transform
            rayDir = {dot(camera.matrix.vx, xyz_position),
                      dot(camera.matrix.vy, xyz_position),
                      dot(camera.matrix.vz, xyz_position)};
        }

        optixTrace(optixLaunchParams.traversable,
                   camera.position,
                   rayDir,
                   0.f,   // tmin
                   1e20f, // tmax
                   0.0f,  // rayTime
                   OptixVisibilityMask(255),
                   OPTIX_RAY_FLAG_DISABLE_ANYHIT, //OPTIX_RAY_FLAG_NONE,
                   RADIANCE_RAY_TYPE,             // SBT offset
                   RAY_TYPE_COUNT,                // SBT stride
                   RADIANCE_RAY_TYPE,             // missSBTIndex
                   u0, u1);
        pixelColor += prd.pixelColor;
    }

    const int r = int(255.99f * min(pixelColor.x / numPixelSamples, 1.f));
    const int g = int(255.99f * min(pixelColor.y / numPixelSamples, 1.f));
    const int b = int(255.99f * min(pixelColor.z / numPixelSamples, 1.f));

    // convert to 32-bit rgba value (we explicitly set alpha to 0xff
    // to make stb_image_write happy ...
    const uint32_t rgba = 0xff000000 | (r << 0) | (g << 8) | (b << 16);

    // and write to frame buffer ...
    const uint32_t fbIndex = ix + iy * optixLaunchParams.frame.size.x;
    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
}